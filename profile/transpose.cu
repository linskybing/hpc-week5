#include <hip/hip_runtime.h>
#include <iostream>

#define ROWS 8192
#define COLS 4096

__global__ void transpose(float* out, const float* in, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        out[col * rows + row] = in[row * cols + col]; // transpose
    }
}

int main() {
    size_t size_in = ROWS * COLS;
    size_t bytes_in = size_in * sizeof(float);

    // Host allocation
    float* h_in = new float[size_in];
    float* h_out = new float[size_in];

    // Initialize input
    for (size_t i = 0; i < size_in; i++)
        h_in[i] = float(i);

    // Device allocation
    float *d_in, *d_out;
    hipMalloc(&d_in, bytes_in);
    hipMalloc(&d_out, bytes_in);

    // Copy to GPU
    hipMemcpy(d_in, h_in, bytes_in, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((COLS + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ROWS + threadsPerBlock.y - 1) / threadsPerBlock.y);

    transpose<<<numBlocks, threadsPerBlock>>>(d_out, d_in, ROWS, COLS);
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(h_out, d_out, bytes_in, hipMemcpyDeviceToHost);

    // Verify
    bool correct = true;
    for (int r = 0; r < ROWS; ++r) {
        for (int c = 0; c < COLS; ++c) {
            float expected = h_in[r * COLS + c];
            float got = h_out[c * ROWS + r];  // transpose
            if (fabs(expected - got) > 1e-5) {
                std::cout << "Mismatch at (" << r << "," << c << "): "
                          << got << " != " << expected << "\n";
                correct = false;
                break;
            }
        }
        if (!correct) break;
    }

    if (correct)
        std::cout << "Transpose verification PASSED!\n";
    else
        std::cout << "Transpose verification FAILED!\n";

    // Free memory
    hipFree(d_in);
    hipFree(d_out);
    delete[] h_in;
    delete[] h_out;

    return 0;
}