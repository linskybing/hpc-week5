#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>

#define TILE_DIM 32
#define EPSILON 1e-3f
#define ceil(a, b) ((a + b - 1) / b)

__global__ void matMulShared(const float* A, const float* B, float* C, int N, int K) {
    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;

    float sum = 0.0f;

    for(int t = 0; t < ceil(K, TILE_DIM); t++) {
        int tiledCol = t * TILE_DIM + threadIdx.x;
        int tiledRow = t * TILE_DIM + threadIdx.y;

        As[threadIdx.y][threadIdx.x] = (row < N && tiledCol < K) ? A[row * K + tiledCol] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (tiledRow < K && col < N) ? B[tiledRow * N + col] : 0.0f;

        __syncthreads();

        for(int i=0; i<TILE_DIM; i++)
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];

        __syncthreads();
    }

    if(row < N && col < N)
        C[row * N + col] = sum;
}

void matMulCPU(const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C, int N, int K) {
    for(int i = 0; i < N;i++)
        for(int j = 0; j < N; j++) {
            float sum = 0.0f;
            for(int k = 0; k < K; k++)
                sum += A[i * K + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
}

int main() {
    const int N = 1024;
    const int K = TILE_DIM;

    std::vector<float> h_A(N * K), h_B(K * N), h_C_cpu(N * N), h_C_gpu(N * N);
    for(int i = 0; i< N * K; i++) h_A[i] = static_cast<float>(rand() % 10);
    for(int i=0; i < K * N; i++) h_B[i] = static_cast<float>(rand() % 10);

    matMulCPU(h_A, h_B, h_C_cpu, N, K);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * K *sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A.data(), N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid(ceil(N, TILE_DIM), ceil(N, TILE_DIM));

    matMulShared<<<grid, block>>>(d_A, d_B, d_C, N, K);
    hipDeviceSynchronize();

    hipMemcpy(h_C_gpu.data(), d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    int error_count = 0;
    for(int i = 0; i < N * N; i++) {
        float diff = fabs(h_C_gpu[i]-h_C_cpu[i]);
        float denom = fabs(h_C_cpu[i])>1e-6f ? fabs(h_C_cpu[i]) : 1.0f;
        if(diff / denom > EPSILON / 1000) {
            if(error_count<10)
                std::cout << "Mismatch at " << i 
                          << ": GPU=" << h_C_gpu[i] 
                          << ", CPU=" << h_C_cpu[i] << "\n";
            error_count++;
        }
    }

    if(error_count == 0)
        std::cout << "GPU result matches CPU reference.\n";
    else
        std::cout << "Total mismatches: " << error_count << "\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
